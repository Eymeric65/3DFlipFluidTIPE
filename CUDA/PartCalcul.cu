#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include ""
#include "CLASSsim/ParticleClass.h"
#include <hip/hip_runtime_api.h> 

__global__ void EulerIntegration(unsigned int partcount,float3* pos, float3* vit, float tstep)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = index; i < partcount; i += stride)
    {
        pos[index].x += vit[index].x * tstep;
        pos[index].y += vit[index].y * tstep;
        pos[index].z += vit[index].z * tstep;
    }
}

extern "C"
void eulercompute(ParticleSystem * partEngine)
{
    EulerIntegration<<<1000,1024>>>(partEngine->PartCount,
                                    partEngine->Partpos,
                                    partEngine->Partvit,
                                    partEngine->TimeStep);
}